#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <ctime>
#include <cstdint>
#include <thrust/reduce.h>
#include <hip/hip_runtime.h>
using namespace std;


__device__ int binarySearch(int* arr, int l, int r, int x)
    {

         while (l <= r)
        {
        int m = (l+r)/2;
 
        
        if (arr[m] == x)
            return m;
 
        
        if (arr[m] < x)
            l = m + 1;
 
        
        else
            r = m - 1;
        }
 

    return -1;
    }

/*__device__ int index; 

__global__ void arrfind(int* adjlist, int start , int end,int entries,int find)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;

  if(threadID  < entries)
  {

     if( adjlist[threadID] == find )
    {
      index = threadID;
    }


  }
}*/


__global__ void Tricount(int* beginposition , int* adjlist ,int* d_counts,int* adjver ,int vertices , int entries)
{
	
	int adjindex = blockIdx.x * blockDim.x + threadIdx.x;
	
  //int vertex =0 ;

  // INDENTIFY WHICH VERTEX THE THREAD IS 
  
	if( adjindex < entries )
	{	

    int vertex1 = adjlist[adjindex];

    int sizeofarray1 = beginposition[ vertex1+1 ]-beginposition[ vertex1 ];

    if( adjlist[adjindex]+1 == vertices)
    {
        sizeofarray1 = entries-beginposition[vertex1];
    }


    int vertex2 = adjver[adjindex];

    int sizeofarray2 = beginposition[vertex2+1]-beginposition[vertex2];

    if( vertex2+1 == vertices)
    {
        sizeofarray2 = entries-beginposition[vertex2];
    }

    int posofelement = beginposition[ adjlist[adjindex] ] ;

    for(int i = 0 ; i  < sizeofarray1 ; i++)
    {

      int find = adjlist[ posofelement + i ];

      int result = binarySearch (adjlist ,beginposition[vertex2] , beginposition[vertex2] + sizeofarray2 - 1 ,find);

      if(result != -1)
      {
        //printf("found an triangle with vertex %d and vertex %d with vertex %d \n",adjlist[adjindex],vertex2,find);
        d_counts[adjindex] = d_counts[adjindex] + 1;
      }

    }


  }

}


int mmioread(int* adjlist , int* beginposition) {
  string line;
  string file1 = "facebook_combined_adj.tsv";
  ifstream myfile (file1);

  cout << endl;
  cout  << " reading " << file1 << " ... " <<endl;
  cout <<endl;
  long linecount =0;
   // 0 - adjlist 1 - vertex 2 - N/A 
  
  beginposition[0] = 0;
  long adjlistpos = 0;
  long beginlistpos = 1;

  long prevnum = 0;
  if (myfile.is_open())
  {
    while ( getline (myfile,line) )
    { 
	  istringstream buf(line);

      
      		long type =0;
          for(string word; buf >> word; )
          {	
          	
            if( type == 0 ) // add adjlist
            {
                adjlist[adjlistpos] = stoi(word);
                adjlistpos++;   
                type++; 
            }

            else if( type == 1 ) // add begin pos
            {   

                if(prevnum != stoi(word) )
                {
                	if (prevnum+1 != stoi(word) )
            		{	
                  //printf("now is %d but before was %d\n",stoi(word),prevnum );
            			for(int a = 0 ; a <stoi(word)-prevnum-1 ; a++)
            			{
            				beginposition[beginlistpos] = adjlistpos-1;
                    //printf("IN \n" );
                    //printf("putting %d at beginpos %d\n",int(adjlistpos-1),int(beginlistpos));
            				beginlistpos++;
            			}
                  
            			
            		}	
                  
                  beginposition[beginlistpos] = adjlistpos-1;

                  beginlistpos++;

                  prevnum = stoi(word);
                }

                type++;
            }
            else if (type == 2)
            	type++;

           	

          	//forcount++;

          }
        
      
      linecount++;
    }
    myfile.close();
  }

  else cout << "Unable to open file"; 

  return 1;
};


int main(){

int vertices = 4040;
int entries = 176468;

int* h_beginposition= new int[vertices];
int* h_adjlist= new int[entries];
int* h_adjvertex= new int[entries];
int* h_count = new int [entries];
//h_count=(int *) malloc(1*sizeof(int));

int* d_begin;
int* d_adj;
int* d_counts;
int* d_adjvertex;

cout <<"Converting MMIO to array form..." <<endl;

clock_t startTime = clock();

mmioread(h_adjlist,h_beginposition);

int pos =0;

for(int x = 1 ; x < vertices ; x++)
{
  int size = h_beginposition[x+1] - h_beginposition[x];
  //printf("%d \n ",size);
  if(x+1 == vertices)
    size = entries-h_beginposition[x];


  for(int y = 0 ; y < size ; y++)
  {
    h_adjvertex[pos] = x;
    pos++;
  }
}

//printf("pos is %d is  %d \n",h_adjlist[718264] ,h_adjvertex[718264]);

//printf("last is %d \n", h_beginposition[4]);
/*
printf("adjlist consist of");
for(int a = 0 ; a < entries ; a++)
	printf(" %d ", h_adjlist[a]);

printf("\n");

printf("bp consist of");
for(int a = 0 ; a < vertices ; a++)
	printf(" %d ", h_beginposition[a]);

printf("\n");*/

double secondsPassed = (clock() - startTime) / CLOCKS_PER_SEC;

cout <<"Transform complete : "<< secondsPassed << " seconds have passed" << endl;

cout <<"Allocating space on GPU and transfer data..."<< endl;
hipMalloc(&d_begin, vertices*sizeof(int)); 
hipMalloc(&d_adj, entries*sizeof(int));
hipMalloc(&d_adjvertex, entries*sizeof(int));  
hipMalloc((void**)&d_counts, entries*sizeof(int));

//hipMemset((void*)d_counts,0,10*sizeof(int));

hipMemcpy(d_begin, h_beginposition, vertices*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_adj, h_adjlist, entries*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_adjvertex, h_adjvertex, entries*sizeof(int), hipMemcpyHostToDevice);


int blocks = (entries/1024)+1;

cout << "Now counting Triangles" <<endl;

Tricount<<<blocks, 1024>>>(d_begin,d_adj,d_counts,d_adjvertex,vertices,entries);

hipMemcpy(h_count,d_counts,entries*sizeof(int),hipMemcpyDeviceToHost);
cout << "Done..." <<endl; 
cout << "Done with MEMCOPY...Now counting" <<endl;

int result = thrust::reduce(h_count, h_count+ entries);
 
printf("answer : %d \n",result/6);




hipFree(d_begin);

hipFree(d_adj);

hipFree(d_counts);
//hipDeviceReset();

//3686467

}
