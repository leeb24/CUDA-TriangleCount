#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <ctime>
#include <cstdint>
#include <thrust/reduce.h>
#include <hip/hip_runtime.h>
using namespace std;


__device__ int binarySearch(int* arr, int l, int r, int x)
    {

         while (l <= r)
        {
        int m = (l+r)/2;
 
        
        if (arr[m] == x)
            return m;
 
        
        if (arr[m] < x)
            l = m + 1;
 
        
        else
            r = m - 1;
        }
 

    return -1;
    }


__global__ void Tricount(int* beginposition , int* adjlist ,int* d_counts,int* adjver ,int vertices , int entries)
{
	
	int adjindex = blockIdx.x * blockDim.x + threadIdx.x;
	
  int vertex =0 ;

  // INDENTIFY WHICH VERTEX THE THREAD IS 
  
	if( adjindex < entries )
	{	

    vertex = adjver[adjindex];

    

    int initial_find = 0;
  
  //FIND ITSELF IN ADJLIST
    for(int a = vertex + 1 ; a < vertices ; a++)
    {

       int sizeofarray1 = beginposition[a+1]-beginposition[a];

       if( a+1 == vertices)
          sizeofarray1 = entries-beginposition[a];


       initial_find = binarySearch(adjlist , beginposition[a] , beginposition[a] + sizeofarray1 -1 , adjlist[adjindex]);

       
       
      if(initial_find != -1)// IF FOUND, FIND VERTEX IN VERTEX2 ADJ
      {

        int vertex2 = adjver[initial_find];

        int sizeofarray = beginposition[vertex2+1]-beginposition[vertex2];

        if(vertex2+1 == vertices)
            sizeofarray = entries-beginposition[vertex2];

        int last_connection = binarySearch(adjlist,beginposition[vertex2],beginposition[vertex2] + sizeofarray -1,vertex);
        
        if(last_connection != -1)//FOUND TRIANGLE
        {
          //atomicAdd(&d_counts[0],1);
          //printf(" %d ",d_counts[0]);
          d_counts[adjindex] = d_counts[adjindex] + 1;
        }
        
      }


    }

	}
  

}


int mmioread(int* adjlist , int* beginposition) {
  string line;
  ifstream myfile ("email-EuAll_adj.tsv");
  long linecount =0;
   // 0 - adjlist 1 - vertex 2 - N/A 
  
  beginposition[0] = 0;
  long adjlistpos = 0;
  long beginlistpos = 1;

  long prevnum = 0;
  if (myfile.is_open())
  {
    while ( getline (myfile,line) )
    { 
	  istringstream buf(line);

      
      		long type =0;
          for(string word; buf >> word; )
          {	
          	
            if( type == 0 ) // add adjlist
            {
                adjlist[adjlistpos] = stoi(word);
                adjlistpos++;   
                type++; 
            }

            else if( type == 1 ) // add begin pos
            {   

                if(prevnum != stoi(word) )
                {
                	if (prevnum+1 != stoi(word) )
            		{	
                  //printf("now is %d but before was %d\n",stoi(word),prevnum );
            			for(int a = 0 ; a <stoi(word)-prevnum-1 ; a++)
            			{
            				beginposition[beginlistpos] = adjlistpos-1;
                    //printf("IN \n" );
                    //printf("putting %d at beginpos %d\n",int(adjlistpos-1),int(beginlistpos));
            				beginlistpos++;
            			}
                  
            			
            		}	
                  
                  beginposition[beginlistpos] = adjlistpos-1;

                  beginlistpos++;

                  prevnum = stoi(word);
                }

                type++;
            }
            else if (type == 2)
            	type++;

           	

          	//forcount++;

          }
        
      
      linecount++;
    }
    myfile.close();
  }

  else cout << "Unable to open file"; 

  return 1;
};


int main(){

int vertices = 265215;
int entries = 728962;

int* h_beginposition= new int[vertices];
int* h_adjlist= new int[entries];
int* h_adjvertex= new int[entries];
int* h_count = new int [entries];
//h_count=(int *) malloc(1*sizeof(int));

int* d_begin;
int* d_adj;
int* d_counts;
int* d_adjvertex;

cout <<"Converting MMIO to array form..." <<endl;

clock_t startTime = clock();

mmioread(h_adjlist,h_beginposition);

int pos =0;

for(int x = 1 ; x < vertices ; x++)
{
  int size = h_beginposition[x+1] - h_beginposition[x];
  //printf("%d \n ",size);
  if(x+1 == vertices)
    size = entries-h_beginposition[x];


  for(int y = 0 ; y < size ; y++)
  {
    h_adjvertex[pos] = x;
    pos++;
  }
}

//printf("pos is %d is  %d \n",h_adjlist[718264] ,h_adjvertex[718264]);

//printf("last is %d \n", h_beginposition[4]);
/*
printf("adjlist consist of");
for(int a = 0 ; a < entries ; a++)
	printf(" %d ", h_adjlist[a]);

printf("\n");

printf("bp consist of");
for(int a = 0 ; a < vertices ; a++)
	printf(" %d ", h_beginposition[a]);

printf("\n");*/

double secondsPassed = (clock() - startTime) / CLOCKS_PER_SEC;

cout <<"Transform complete : "<< secondsPassed << " seconds have passed" << endl;

cout <<"Allocating space on GPU and transfer data..."<< endl;
hipMalloc(&d_begin, vertices*sizeof(int)); 
hipMalloc(&d_adj, entries*sizeof(int));
hipMalloc(&d_adjvertex, entries*sizeof(int));  
hipMalloc((void**)&d_counts, entries*sizeof(int));

//hipMemset((void*)d_counts,0,10*sizeof(int));

hipMemcpy(d_begin, h_beginposition, vertices*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_adj, h_adjlist, entries*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_adjvertex, h_adjvertex, entries*sizeof(int), hipMemcpyHostToDevice);


int blocks = (entries/1024)+1;

cout << "Now counting Triangles" <<endl;

Tricount<<<blocks, 1024>>>(d_begin,d_adj,d_counts,d_adjvertex,vertices,entries);

cout << "Done..." <<endl; 

hipMemcpy(h_count,d_counts,entries*sizeof(int),hipMemcpyDeviceToHost);

cout << "Done with MEMCOPY...Now counting" <<endl;

int result = thrust::reduce(h_count, h_count+ entries);
 
printf("answer : %d \n",result/3);




hipFree(d_begin);

hipFree(d_adj);

hipFree(d_counts);
//hipDeviceReset();

//3686467

}
