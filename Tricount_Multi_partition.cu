#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <ctime>
#include <cstdint>
#include <thrust/reduce.h>
#include <hip/hip_runtime.h>
using namespace std;


__device__ int binarySearch(int* arr, int l, int r, int x)
    {

         while (l <= r)
        {
        int m = (l+r)/2;
 
        
        if (arr[m] == x)
            return m;
 
        
        if (arr[m] < x)
            l = m + 1;
 
        
        else
            r = m - 1;
        }
 

    return -1;
    }

/*__device__ int index; 

__global__ void arrfind(int* adjlist, int start , int end,int entries,int find)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;

  if(threadID  < entries)
  {

     if( adjlist[threadID] == find )
    {
      index = threadID;
    }


  }
}*/


__global__ void Tricount(int* beginposition , int* graphpartition , int* d_counts , int* adjver , int vertices , int entries,int partitionvertex,int partitionedge,int* adjlist,int part)
{
	
	int thread = blockIdx.x * blockDim.x + threadIdx.x;

  if(thread < partitionedge ) // limit thread to how many edges 
  {

      
    	if( part ==2 ) //SECOND PARTITON
    	{

    		if(graphpartition[thread] > partitionvertex ) // if the first vertex is whithin the partition (POSSUIBEL ERROR)
    		{
    			int vertex1 = graphpartition[thread];

    			int sizeofarray1 = beginposition[ vertex1+1 ]- beginposition[ vertex1 ];

      		if( graphpartition[thread]+1 == vertices) //vertices has to be changed too
      		{
          			sizeofarray1 = entries-beginposition[vertex1];
      		}

      		int vertex2 = adjver[thread];

      		int sizeofarray2 = beginposition[ vertex2+1 ]-beginposition[ vertex2 ];

      		if( vertex2+1 == vertices)
      		{
          		sizeofarray2 = entries-beginposition[vertex2];
      		}

      		int posofelement = beginposition[vertex1];

    			for(int i = 0 ; i  < sizeofarray1 ; i++)
        	{

      			int find = graphpartition[ posofelement + i ];

      			int result = binarySearch (adjlist ,beginposition[vertex2] , beginposition[vertex2] + sizeofarray2 - 1 ,find);

      			if(result != -1)
      			{
        			//printf("found an triangle with vertex %d and vertex %d with vertex %d \n",adjlist[adjindex],vertex2,find);
        			d_counts[thread] = d_counts[thread] + 1;
              //printf("I found a triangle");
          	}

        	}
    		}

    		
    	}



    	else //FIRST PARTITION
    	{
        
    		if(graphpartition[thread] <= partitionvertex ) // if the first vertex is whithin the partition
    		{ 
    			int vertex1 = graphpartition[thread];

        
    			int sizeofarray1 = beginposition[ vertex1+1 ]- beginposition[ vertex1 ];

      		if( graphpartition[thread]+1 == partitionvertex +1) //vertices has to be changed too
      		{
          			sizeofarray1 = entries-beginposition[vertex1];
      		}

      		int vertex2 = adjver[thread];

      		int sizeofarray2 = beginposition[vertex2+1]-beginposition[vertex2];

      		if( vertex2+1 == partitionvertex +1)
      		{
          		sizeofarray2 = entries-beginposition[vertex2];
      		}

      		int posofelement = beginposition[vertex1];

    			for(int i = 0 ; i  < sizeofarray1 ; i++)
        	{

      			int find = graphpartition[ posofelement + i ];

      			int result = binarySearch (adjlist ,beginposition[vertex2] , beginposition[vertex2] + sizeofarray2 - 1 ,find);//adjust (Find Intersection)

      			if(result != -1)
      			{
        			//printf("found an triangle with vertex %d and vertex %d with vertex %d \n",adjlist[adjindex],vertex2,find);
        			d_counts[thread] = d_counts[thread] + 1;
              //printf("I found a triangle");
      			}

        	}
    		}

    	}

  }
	
}


int mmioread(int* adjlist , int* beginposition) {
  string line;
  string file1 = "amazon0312_adj.tsv";
  ifstream myfile (file1);

  cout << endl;
  cout  << " reading " << file1 << " ... " <<endl;
  cout <<endl;
  long linecount =0;
   // 0 - adjlist 1 - vertex 2 - N/A 
  
  beginposition[0] = 0;
  long adjlistpos = 0;
  long beginlistpos = 1;

  long prevnum = 0;
  if (myfile.is_open())
  {
    while ( getline (myfile,line) )
    { 
	  istringstream buf(line);

      
      		long type =0;
          for(string word; buf >> word; )
          {	
          	
            if( type == 0 ) // add adjlist
            {
                adjlist[adjlistpos] = stoi(word);
                adjlistpos++;   
                type++; 
            }

            else if( type == 1 ) // add begin pos
            {   

                if(prevnum != stoi(word) )
                {
                	if (prevnum+1 != stoi(word) )
            		{	
                  //printf("now is %d but before was %d\n",stoi(word),prevnum );
            			for(int a = 0 ; a <stoi(word)-prevnum-1 ; a++) //Parsing Error Fix
            			{
            				beginposition[beginlistpos] = adjlistpos-1;
            				beginlistpos++;
            			}
                  
            			
            		}	
                  
                  beginposition[beginlistpos] = adjlistpos-1;

                  beginlistpos++;

                  prevnum = stoi(word);
                }

                type++;
            }
            else if (type == 2)
            	type++;

           	

          	//forcount++;

          }
        
      
      linecount++;
    }
    myfile.close();
  }

  else cout << "Unable to open file"; 

  return 1;
};


int main(){

int vertices = 400728;
int entries = 4699738;

int* h_beginposition= new int[vertices];
int* h_adjlist= new int[entries];
int* h_adjvertex= new int[entries];
int* h_count = new int [entries];
int* h_count2 = new int [entries];

int* d_begin;
int* d_adj;
int* d_counts;
int* d_counts2;
int* d_adjvertex;

cout <<"Converting MMIO to array form..." <<endl;

clock_t startTime = clock();

mmioread(h_adjlist,h_beginposition);

int pos =0;

for(int x = 1 ; x < vertices ; x++)
{
  int size = h_beginposition[x+1] - h_beginposition[x];
  //printf("%d \n ",size);
  if( x+1 == vertices )
    size = entries-h_beginposition[x];


  for(int y = 0 ; y < size ; y++)
  {
    h_adjvertex[pos] = x;
    pos++;
  }
}

//*****************************************************************************************************

int partition = vertices/2;
cout << "partition vertex is : " << partition << endl;


int sizeofpart1 = h_beginposition[partition+1];
cout << "sizeof partion is : " << sizeofpart1 << endl;

int* h_graphpartition1 = new int[ sizeofpart1 ];
int* h_graphpartition2 = new int[ entries - sizeofpart1 ];

int* h_adjver1 = new int[h_beginposition[partition+1]];
int* h_adjver2 = new int[entries - sizeofpart1];
 
int* d_graphpartition1;
int* d_graphpartition2;
int* d_adjver1;
int* d_adjver2;

//*****************************************************************************************************
//PARTITION DATASETS 
//**************************************************************************************************

for(int i = 0 ; i < h_beginposition[partition+1] ; i++)
{
	 h_graphpartition1[i] = h_adjlist[i];
	 h_adjver1[i] = h_adjvertex[i];

}
for(int i = 0 ; i < entries - (h_beginposition[partition+1]) ; i++)
{
	 h_graphpartition2[i] = h_adjlist[ i + h_beginposition[partition+1] ];
	 h_adjver2[i] = h_adjvertex[ i + h_beginposition[partition+1] ];
}

cout <<"last is : " << h_graphpartition2[entries - (h_beginposition[partition+1])-1] <<endl;

int checkvertex = h_adjvertex[ h_beginposition[partition+1] -1 ]; //UPTO WHERE TO COPY BP

int* h_BP1 = new int[checkvertex+1]; 
int* h_BP2 = new int[ vertices ];


for(int i = 0 ; i < (checkvertex+1) ; i++)
{
	h_BP1[i] = h_beginposition[i];
}
for(int i =0 ; i < vertices-1 ; i++)
{	
	if(i>checkvertex)
		h_BP2[i] = h_beginposition[i]-h_beginposition[checkvertex+1]; //convert to partition
}
h_BP2[3] =0;
//********************************************************************************************************
//DEBUG SESSION 
//printf("pos is %d is  %d \n",h_adjlist[718264] ,h_adjvertex[718264]);

//printf("last is %d \n", h_beginposition[4]);
/*
printf("adjlist consist of");
for(int a = 0 ; a < entries ; a++)
	printf(" %d ", h_adjlist[a]);

printf("\n");

printf("bp consist of");
for(int a = 0 ; a < vertices ; a++)
	printf(" %d ", h_beginposition[a]);

printf("\n");*/
//********************************************************************************************************
//MEMORY ALLOCATION ON DEVICE & MEMORY TRANSFER TO DEVICE

double secondsPassed = (clock() - startTime) / CLOCKS_PER_SEC;

cout <<"Transform complete : "<< secondsPassed << " seconds have passed" << endl;

cout <<"Allocating space on GPU and transfer data..."<< endl;

cout <<"index 2 value is " << h_graphpartition1[3]<<endl;

hipMalloc(&d_begin, vertices*sizeof(int)); 
hipMalloc(&d_adj, entries*sizeof(int));
//hipMalloc(&d_adjvertex, entries*sizeof(int));  
hipMalloc((void**)&d_counts, entries*sizeof(int));
hipMalloc((void**)&d_counts2, entries*sizeof(int));

hipMalloc(&d_graphpartition1,sizeofpart1*sizeof(int));
hipMalloc(&d_graphpartition2,(entries-sizeofpart1)*sizeof(int));

hipMalloc(&d_adjver1,sizeofpart1*sizeof(int));
hipMalloc(&d_adjver2,(entries-sizeofpart1)*sizeof(int));
//hipMemset((void*)d_counts,0,10*sizeof(int));

//**********************************************************************************************************************

hipMemcpy(d_begin, h_beginposition, vertices*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_adj, h_adjlist, entries*sizeof(int), hipMemcpyHostToDevice);
//hipMemcpy(d_adjvertex, h_adjvertex, entries*sizeof(int), hipMemcpyHostToDevice);

hipMemcpy(d_graphpartition1,h_graphpartition1,sizeofpart1*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_graphpartition2,h_graphpartition2,(entries-sizeofpart1)*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_adjver1,h_adjver1,sizeofpart1*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_adjver2,h_adjver2,(entries-sizeofpart1)*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_counts2,h_count2,(entries-sizeofpart1)*sizeof(int),hipMemcpyHostToDevice);

int blocks = (entries/1024)+1;
cout << "Now counting Triangles" <<endl;

Tricount<<<blocks, 1024>>>(d_begin ,d_graphpartition1 ,d_counts ,d_adjver1 ,vertices , entries,partition,sizeofpart1,d_adj,1);
Tricount<<<blocks, 1024>>>(d_begin ,d_graphpartition2 ,d_counts2 ,d_adjver2 ,vertices , entries,partition,sizeofpart1,d_adj,2);

hipMemcpy(h_count,d_counts,entries*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(h_count2,d_counts2,entries*sizeof(int),hipMemcpyDeviceToHost);
cout << "Done..." <<endl; 
cout << "Done with MEMCOPY...Now counting" <<endl;

int result = thrust::reduce(h_count, h_count+ entries);
int result2 = thrust::reduce(h_count2, h_count2+ entries);
 
printf("First Partition Triangles >>>>> %d \n",result/6);
printf("Second Partition Triangles >>>>> %d \n",result2/6);

printf("Total number is %d\n",(result2+result)/6 );

hipFree(d_begin);

hipFree(d_adj);

hipFree(d_counts);

hipFree(d_graphpartition1);
hipFree(d_graphpartition2);
hipFree(d_adjver1);
hipFree(d_adjver2);
//hipDeviceReset();

//3686467

}
