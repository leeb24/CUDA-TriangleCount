#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <ctime>
#include <cstdint>
#include <algorithm>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
using namespace std;


struct node
{
  public:

    int VertexID;
    int Degree;
    int part;
    int* adj= new int[200364];
    int* d_adj;
    node()
    {
      hipMalloc(&d_adj,200364*sizeof(int));
    }
    void sendGPU()
    {
      hipMemcpy(d_adj,adj,200364*sizeof(int),hipMemcpyHostToDevice);
    }
    
};

__device__ __host__ bool cmp(const node node1 ,const node node2)
{
  if(node1.VertexID != node2.VertexID)
    return node1.Degree > node2.Degree;
}

__device__ __host__ bool cmp2(const node node1 ,const node node2)
{
    return node1.VertexID < node2.VertexID;
}


__device__ int binarySearch(int* arr, int l, int r, int x)
    {

        while (l <= r)
        {
          int m = (l+r)/2;
 
        
          if (arr[m] == x)
            return m;
 
        
          if (arr[m] < x)
            l = m + 1;
 
        
          else
            r = m - 1;
        }
 

        return -1;
      }



__global__ void Tricount2(int* beginposition , int* adjlist ,int* d_counts,int* adjver ,int vertices , int entries , int* degree_array, int* d_partition)
{
  
  int adjindex = blockIdx.x * blockDim.x + threadIdx.x;
  
  ///SECOND PARTITION

  // INDENTIFY WHICH VERTEX THE THREAD IS 
  
  if( adjindex < entries  && adjindex >= (entries/2) + 1 )
  { 

    int vertex1 = adjlist[adjindex];

    if (d_partition[vertex1] == 2)
    {
        int sizeofarray1 = degree_array[vertex1];


        int vertex2 = adjver[adjindex];
        if(d_partition[vertex2] == 2)
        {
          int sizeofarray2 = degree_array[vertex2];

        
          int posofelement = beginposition[ vertex1 ] ;

          for(int i = 0 ; i  < sizeofarray1 ; i++)
          {

            int find = adjlist[ posofelement + i ];

            int result = binarySearch (adjlist ,beginposition[vertex2] , beginposition[vertex2] + sizeofarray2 - 1 ,find);

            if(result != -1)
            {
              //printf("found an triangle with vertex %d and vertex %d with vertex %d \n",adjlist[adjindex],vertex2,find);
              d_counts[adjindex] = d_counts[adjindex] + 1;
            }

           }
      }
    }
    //
  }

}

//FIRST PARTITION 

__global__ void Tricount(int* beginposition , int* adjlist ,int* d_counts,int* adjver ,int vertices , int entries,int* degree_array,int* d_partition)
{
	
	int adjindex = blockIdx.x * blockDim.x + threadIdx.x;
	
  //int vertex =0 ;

  // INDENTIFY WHICH VERTEX THE THREAD IS 
  
	if( adjindex < entries  && adjindex < (entries/2) + 1)
	{	
    //("In first Kernel\n");
    int vertex1 = adjlist[adjindex];

    if (d_partition[vertex1] == 1) //
    {
        int sizeofarray1 = degree_array[vertex1];

        int vertex2 = adjver[adjindex];
        if(d_partition[vertex2] == 1)
        {
          int sizeofarray2 = degree_array[vertex2];

          int posofelement = beginposition[ vertex1 ] ;

          for(int i = 0 ; i  < sizeofarray1 ; i++)
          {

            int find = adjlist[ posofelement + i ];

            int result = binarySearch (adjlist ,beginposition[vertex2] , beginposition[vertex2] + sizeofarray2 - 1 ,find);

            if(result != -1)
            {
              //printf("found an triangle with vertex %d and vertex %d with vertex %d \n",adjlist[adjindex],vertex2,find);
              d_counts[adjindex] = d_counts[adjindex] + 1;
            }

          }
        }
    }
    //
  }

}


int mmioread(int* adjlist , int* beginposition) {
  string line;
  string file1 = "email-Enron_adj.tsv";
  ifstream myfile (file1);

  cout << endl;
  cout  << " reading " << file1 << " ... " <<endl;
  cout <<endl;
  long linecount =0;
   // 0 - adjlist 1 - vertex 2 - N/A 
  
  beginposition[0] = 0;
  long adjlistpos = 0;
  long beginlistpos = 1;

  long prevnum = 0;
  if (myfile.is_open())
  {
    while ( getline (myfile,line) )
    { 
	  istringstream buf(line);

      
      		long type =0;
          for(string word; buf >> word; )
          {	
          	
            if( type == 0 ) // add adjlist
            {
                adjlist[adjlistpos] = stoi(word);
                adjlistpos++;   
                type++; 
            }

            else if( type == 1 ) // add begin pos
            {   

                if(prevnum != stoi(word) )
                {
                	if (prevnum+1 != stoi(word) )
            		{	
                  //printf("now is %d but before was %d\n",stoi(word),prevnum );
            			for(int a = 0 ; a <stoi(word)-prevnum-1 ; a++)
            			{
            				beginposition[beginlistpos] = adjlistpos-1;
                    //printf("IN \n" );
                    //printf("putting %d at beginpos %d\n",int(adjlistpos-1),int(beginlistpos));
            				beginlistpos++;
            			}
                  
            			
            		}	
                  
                  beginposition[beginlistpos] = adjlistpos-1;

                  beginlistpos++;

                  prevnum = stoi(word);
                }

                type++;
            }
            else if (type == 2)
            	type++;

           	

          	//forcount++;

          }
        
      
      linecount++;
    }
    myfile.close();
  }

  else cout << "Unable to open file"; 

  return 1;
};


int main(){

int vertices = 36693;
int entries = 367662;

int* h_beginposition= new int[vertices];
int* h_adjlist= new int[entries];
int* h_adjvertex= new int[entries];
int* h_count = new int [entries];
int* h_count2 = new int[entries];
int* h_degrees = new int [vertices];
int* h_partition = new int [entries];
//h_count=(int *) malloc(1*sizeof(int));

int* d_begin;
int* d_adj;
int* d_counts;
int* d_counts2;
int* d_adjvertex;
int* d_degrees;
int* d_partition;

cout <<"Converting MMIO to array form..." <<endl;

clock_t startTime = clock();

cout << "hi" <<endl;

mmioread(h_adjlist,h_beginposition);

cout<< "BP before " << h_adjlist[ h_beginposition[10000]] << endl;

cout << "BP IS " << h_beginposition[10000] << endl;
int pos =0;

for(int x = 1 ; x < vertices ; x++)
{
  int size = h_beginposition[x+1] - h_beginposition[x];
  //printf("%d \n ",size);
  if(x+1 == vertices)
    size = entries-h_beginposition[x];


  for(int y = 0 ; y < size ; y++)
  {
    h_adjvertex[pos] = x;
    pos++;
  }
}

//*************************************************************************************************************

int k = 0;

node* node_degree = new node[vertices];



for(int i = 1 ; i < vertices ; i++)
{
  int sizeofarray1 = h_beginposition[ i+1 ]- h_beginposition[ i ];

  for(int j  = 0 ; j < sizeofarray1 ; j++)
  {
      node_degree[i].adj[k] = h_adjlist[ h_beginposition[i] + j ];
      k++;
  }

  k=0;
  node_degree[i].VertexID = i;
  node_degree[i].Degree = sizeofarray1;
  
}

//size of each vertex degrees ( sizeof variable )

for(int i = 0 ; i < vertices ; i++)
{
  h_degrees[i] = node_degree[i].Degree; // in order with 1.2.3. Vertex ID  
}


std::sort( node_degree , node_degree + vertices ,cmp );//  Descending order sort 


//************************REBUILD ADJLIST AND OTHER DEPENDENCIES ***************************

int adjpos = 0;
for(int i = 1; i < vertices ; i++ )
{
  int degrees = node_degree[i].Degree;
  int nodepos = 0;

  h_beginposition[node_degree[i].VertexID] = adjpos;// Where it starts on the adjlist

  while(degrees > nodepos)
  {
    h_adjlist[adjpos] = node_degree[i].adj[nodepos]; //sorted adjlist 

    h_adjvertex[adjpos] = node_degree[i].VertexID; // sorted connected vertex 

    nodepos++;
    adjpos++;
  }

  nodepos = 0;
}


// partition array 

node_degree[0].part = 0;
for(int i = 1 ; i < (vertices/2)+1 ; i++)
{
  node_degree[i].part = 1;
}
for (int i = (vertices/2)+1 ; i < vertices ; i++)
{
  node_degree[i].part = 2;
}

std::sort( node_degree , node_degree + vertices , cmp2 ); //sort by vertex id 



for (int i = 0; i < vertices; i++)
{
  h_partition[i] = node_degree[i].part;
}


/*for (int i = 0; i <100 ; i++)
{
  cout << node_degree[i].part <<endl;
}*/


cout << "im here"<<endl;

//printf("pos is %d is  %d \n",h_adjlist[718264] ,h_adjvertex[718264]);

//printf("last is %d \n", h_beginposition[4]);
/*
printf("adjlist consist of");
for(int a = 0 ; a < entries ; a++)
	printf(" %d ", h_adjlist[a]);

printf("\n");

printf("bp consist of");
for(int a = 0 ; a < vertices ; a++)
	printf(" %d ", h_beginposition[a]);

printf("\n");*/

cout<< "BP after " << h_adjlist[ h_beginposition[10000]] << endl;

cout << "BP IS 2" << h_beginposition[10000] << endl;

double secondsPassed = (clock() - startTime) / CLOCKS_PER_SEC;

cout <<"Transform complete : "<< secondsPassed << " seconds have passed" << endl;

cout <<"Allocating space on GPU and transfer data..."<< endl;
hipMalloc(&d_begin, vertices*sizeof(int)); 
hipMalloc(&d_adj, entries*sizeof(int));
hipMalloc(&d_adjvertex, entries*sizeof(int));  
hipMalloc((void**)&d_counts, entries*sizeof(int));
hipMalloc((void**)&d_counts2, entries*sizeof(int));
hipMalloc(&d_degrees,vertices*sizeof(int));
hipMalloc(&d_partition, entries*sizeof(int));


hipMemcpy(d_begin, h_beginposition, vertices*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_adj, h_adjlist, entries*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_adjvertex, h_adjvertex, entries*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_degrees,h_degrees,vertices*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_partition,h_partition,entries*sizeof(int),hipMemcpyHostToDevice);


int blocks = (entries/1024)+1;

cout << "Now counting Triangles" <<endl;

Tricount<<<blocks, 1024>>>(d_begin,d_adj,d_counts,d_adjvertex,vertices,entries,d_degrees,d_partition);
Tricount2<<<blocks, 1024>>>(d_begin,d_adj,d_counts2,d_adjvertex,vertices,entries,d_degrees,d_partition);

hipMemcpy(h_count,d_counts,entries*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(h_count2,d_counts2,entries*sizeof(int),hipMemcpyDeviceToHost);
cout << "Done..." <<endl; 
cout << "Done with MEMCOPY...Now counting" <<endl;

int result = thrust::reduce(h_count, h_count+ entries);
int result2 = thrust::reduce(h_count2, h_count2+ entries);
 
printf("answer1 : %d \n",result/6);
printf("answer2 ; %d \n",result2/6);

printf("total is : %d \n", (result+result2)/6);



hipFree(d_begin);

hipFree(d_adj);

hipFree(d_counts);

hipFree(d_counts2);
//hipDeviceReset();

//3686467

}
